#include "hip/hip_runtime.h"
﻿//Practica 1   VECTORES

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void idkernel()
{
    printf("threadIdx %d %d %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("blockIdx %d %d %d \n", blockIdx.x, blockIdx.y, blockIdx.z);
    printf("gridDim %d %d %d \n", gridDim.x, gridDim.y, gridDim.z);
}

__global__ void multiplyVectors(int* a, int* b, int* c)
{
    int id = threadIdx.x;
    c[id] = a[id] * b[id];
}

int main()
{
    int nx = 4;
    int ny = 4;
    int nz = 4;

    dim3 blockDim(2, 2, 2);
    dim3 gridDim(nx / blockDim.x, ny / blockDim.y, nz / blockDim.z);

    idkernel << <gridDim, blockDim >> > ();


    const int vectoSize = 3;

    int* dev_a, * dev_b, * dev_c;
    hipMalloc((void**)&dev_a, vectoSize * sizeof(int));
    hipMalloc((void**)&dev_b, vectoSize * sizeof(int));
    hipMalloc((void**)&dev_c, vectoSize * sizeof(int));

    int* phost_a, * phost_b, * phost_c;
    phost_a = (int*)malloc(vectoSize * sizeof(int));
    phost_b = (int*)malloc(vectoSize * sizeof(int));
    phost_c = (int*)malloc(vectoSize * sizeof(int));

    for (int i = 0; i < vectoSize; i++) {
        phost_a[i] = i;
        phost_b[i] = i;
    }

    hipMemcpy(dev_a, phost_a, vectoSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, phost_b, vectoSize * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(vectoSize);
    dim3 grid(1);

    multiplyVectors << < grid, block >> > (dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    hipMemcpy(phost_c, dev_c, vectoSize * sizeof(int), hipMemcpyDeviceToHost);

    printf("\n\nVector A + Vector B = Vector C");
    for (int i = 0; i < vectoSize; i++) {
        printf("\n     %d   +\t%d   =\t%d", phost_a[i], phost_b[i], phost_c[i]);
    }
    hipDeviceReset();
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}