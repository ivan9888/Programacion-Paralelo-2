#include "hip/hip_runtime.h"
﻿//Practica 5 Multiplicar matrices

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

__global__ void dotProduct(int* a, int* b, int* c, int matSize)
{
    int tid = blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z) + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int bid = gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z) + blockIdx.x;
    int gid = tid + bid * threads_per_block;

    c[gid] = 0;

    int col = (int)(gid / matSize) * matSize;
    int row = (int)(gid % matSize);

    printf("\ngid = %d\tcol = %d\trow = %d", gid, col, row);

    for (int i = 0; i < matSize; i++) {
        c[gid] += a[col] * b[row];
        if (gid == 0) {
            printf("\ngid = %d\tcol = %d\trow = %d", gid, col, row);
        }
        col += 1;
        row += matSize;
    }
}

void printMatrix(int* a, int matSize) {
    for (int i = 0; i < matSize * matSize; i++) {
        if (i % matSize == 0) {
            printf("\n");
        }
        printf("\t%d", a[i]);
    }
}

int main()
{
    const int vectorSize = 9;
    const int size = vectorSize * sizeof(int);
    int matSize = 3;
    int* dev_a, * dev_b, * dev_c;

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    int* phost_a, * phost_b, * phost_c;

    phost_a = (int*)malloc(size);
    phost_b = (int*)malloc(size);
    phost_c = (int*)malloc(size);

    for (int i = 0; i < vectorSize; i++) {
        phost_a[i] = i + 1;
        phost_b[i] = i + 1 + vectorSize;
    }

    hipMemcpy(dev_a, phost_a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, phost_b, size, hipMemcpyHostToDevice);

    dim3 blockDim(matSize, matSize);
    dim3 gridDim(1);

    clock_t gpu_start, gpu_stop;

    gpu_start = clock();
    dotProduct << < gridDim, blockDim >> > (dev_a, dev_b, dev_c, matSize);
    hipDeviceSynchronize();

    gpu_stop = clock();
    double cps_gpu = (double)((double)(gpu_stop - gpu_start) / CLOCKS_PER_SEC);
    printf("\n\nExecution Time [ET.GPU]: %4.6f\n\r", cps_gpu);

    hipMemcpy(phost_c, dev_c, size, hipMemcpyDeviceToHost);

    printf("\n\n*****    MATRIX A    *****\n");
    printMatrix(phost_a, matSize);

    printf("\n\n*****    MATRIX B    *****\n");
    printMatrix(phost_b, matSize);

    printf("\n\n*****    MATRIX C    *****\n");
    printMatrix(phost_c, matSize);

    hipDeviceReset();
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}