#include "hip/hip_runtime.h"
﻿//Practica 3 operaciones

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

__global__ void addVectors(int* a, int* b, int* c, int size)
{
    int tid = blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z) + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y * blockDim.z;

    int bid = gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z) + blockIdx.x;
    int gid = tid + bid * threads_per_block;

    if (gid < size) {
        c[gid] = a[gid] + b[gid];
        printf("\n%d) %d + %d = %d", gid, a[gid], b[gid], c[gid]);
    }
}

__global__ void add3Vectors(int* a, int* b, int* c, int* d, int size)
{
    int tid = blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z) + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y * blockDim.z;

    int bid = gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z) + blockIdx.x;
    int gid = tid + bid * threads_per_block;

    if (gid < size) {
        d[gid] = a[gid] + b[gid] + c[gid];
        printf("\n%d) %d + %d + %d = %d", gid, a[gid], b[gid], c[gid], d[gid]);
    }
}

int main()
{
    const int vectorSize = 10000;
    const int size = vectorSize * sizeof(int);
    int* dev_a, * dev_b, * dev_c, * dev_d;

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    hipMalloc((void**)&dev_d, size);

    int* phost_a, * phost_b, * phost_c, * phost_d, * phost_res;

    phost_a = (int*)malloc(size);
    phost_b = (int*)malloc(size);
    phost_c = (int*)malloc(size);
    phost_d = (int*)malloc(size);
    phost_res = (int*)malloc(size);

    for (int i = 0; i < vectorSize; i++) {
        phost_a[i] = rand() % 255;
        phost_b[i] = rand() % 255;
        phost_c[i] = rand() % 255;
        // phost_d[i] = phost_a[i] + phost_b[i];
        phost_d[i] = phost_a[i] + phost_b[i] + phost_c[i];
        // printf("\n%d) %d + %d + %d = %d", i, phost_a[i], phost_b[i], phost_c[i]);
    }

    hipMemcpy(dev_a, phost_a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, phost_b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, phost_c, size, hipMemcpyHostToDevice);

    dim3 blockDim(4, 4, 8);
    dim3 gridDim(4, 4, 5);

    printf("\n********** Add in kernel **********");

    clock_t gpu_start, gpu_stop;

    gpu_start = clock();
    // addVectors << < gridDim, blockDim >> > (dev_a, dev_b, dev_d, size);
    add3Vectors << < gridDim, blockDim >> > (dev_a, dev_b, dev_c, dev_d, size);
    hipDeviceSynchronize();

    gpu_stop = clock();
    double cps_gpu = (double)((double)(gpu_stop - gpu_start) / CLOCKS_PER_SEC);
    printf("\n\nExecution Time [ET.GPU]: %4.6f\n\r", cps_gpu);

    hipMemcpy(phost_res, dev_d, size, hipMemcpyDeviceToHost);

    bool equal = true;
    for (int i = 0; i < vectorSize; i++) {
        if (phost_d[i] != phost_res[i]) {
            equal = false;
            printf("%d", i);
            break;
        }
    }
    printf("\n\n\nBoth are equal? %s\n\n", equal ? "True" : "False");

    hipDeviceReset();
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_d);

    return 0;
}