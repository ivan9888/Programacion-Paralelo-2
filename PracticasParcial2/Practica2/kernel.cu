#include "hip/hip_runtime.h"
﻿/*Tramnsposicion de matrices*/

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <random>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>




#define TILE_DIM 16

__global__ void transpose_no_SM(double* source, double* dest, int size) {
	int i = threadIdx.x + blockIdx.x + blockDim.x;
	int j = threadIdx.y + blockIdx.y + blockDim.y;

	if (i < size && j < size) {
		int dst_idx = i * size + j;
		int src_idx = j * size + i;
		dest[dst_idx] = source[src_idx];
	}
}


__global__ void transpose_shared(double* source, double* dest, int size) {

	__shared__ double tile[TILE_DIM][TILE_DIM + 1];

	//input threads idx

	int i_in = threadIdx.x + blockDim.x + blockIdx.x;
	int j_in = threadIdx.y + blockIdx.y + blockDim.y;

	//index
	int src_idx = j_in * size + i_in;

	//1D index calculation shared memory
	int _1d_index = threadIdx.y * blockDim.x + threadIdx.x;

	//coordinate for transpose matrix
	int i_out = blockIdx.y * blockDim.y + threadIdx.x;
	int j_out = blockIdx.x * blockDim.y + threadIdx.y;

	//output index
	int dst_idx = j_out * size + i_out;

	if(i_in < size && j_in < size) {
		//Load from in array in row major and store to shared memory in row major
		tile[threadIdx.y][threadIdx.x] = source[src_idx];

		//wait untill all the threads load the values
		__syncthreads();

		dest[dst_idx] = tile[threadIdx.x][threadIdx.y];

	}

}

int main() {
	int mat_size = 4096;
	int byte_size = mat_size * mat_size * sizeof(double);

	//hostallocation
	double* mat_input = (double*)malloc(byte_size);
	double* mat_output = (double*)malloc(byte_size);
	memset(mat_output, 0, byte_size);

	//init array
	srand((unsigned)time(NULL));
	for (int i = 0; i < mat_size * mat_size; i++) {
		mat_input[i] = (double)(rand() % 10);
	}

	//allocate device pointers
	//cuda_ptr<double>in_gpu(mat_input, byte_size);
	//cuda_ptr<double>out_gpu(mat_output, byte_size);

	//launch kernel
	int block_size = TILE_DIM;
	int grid_size = (int)ceil((float)mat_size / block_size);//ceil redondea hacia arriba

	dim3 block(block_size, block_size);

	dim3 grid(grid_size, grid_size);

	transpose_shared << <grid, block >> > (mat_input, mat_output, mat_size);
	//CUDA_ERROR_HANDLER(hipDeviceSynchronize());
	//out_gpu.to_host(mat_output, byte_size);


	return 0;
}