#include "hip/hip_runtime.h"
%%cu
#include stdio.h
#include stdlib.h
#include hip/hip_runtime.h

#define N 4  filas
#define M 5  columnas

__global__ void transpose(int input, int output) {
    __shared__ int tile[N][N+1];
    int x = blockIdx.x  N + threadIdx.x;
    int y = blockIdx.y  N + threadIdx.y;
    int index_in = x + y  M;
    x = blockIdx.y  N + threadIdx.x;
    y = blockIdx.x  N + threadIdx.y;
    int index_out = x + y  N;

    for (int i = 0; i  N; i += blockDim.y) {
        tile[threadIdx.y+i][threadIdx.x] = input[index_in+iM+threadIdx.x];
    }
    __syncthreads();
    
    for (int i = 0; i  N; i += blockDim.y) {
        output[index_out+iN+threadIdx.x] = tile[threadIdx.x][threadIdx.y+i];
    }
}

int main() {
    int input[NM] = {1, 2, 3, 4, 5,
                      6, 7, 8, 9, 10,
                      11, 12, 13, 14, 15,
                      16, 17, 18, 19, 20};
    int output[MN];

    int d_input, d_output;
    hipMalloc(&d_input, NMsizeof(int));
    hipMalloc(&d_output, MNsizeof(int));

    hipMemcpy(d_input, input, NMsizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid((M+N-1)N, (N+N-1)N, 1);
    dim3 dimBlock(N, N, 1);

    transposedimGrid, dimBlock(d_input, d_output);

    hipMemcpy(output, d_output, MNsizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    printf(Matriz transpuestan);
    for (int i = 0; i  M; i++) {
        for (int j = 0; j  N; j++) {
            printf(%d , output[iN+j]);
        }
        printf(n);
    }

    return 0;
}
