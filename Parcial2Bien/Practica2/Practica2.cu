#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <random>

#define BLOCK_SIZE 1024
#define N 1000

__global__ void bubbleSort(int *array) {
    __shared__ int sharedArray[BLOCK_SIZE];
    int i, j;

    // Cargar datos en shared memory
    sharedArray[threadIdx.x] = array[blockIdx.x * BLOCK_SIZE + threadIdx.x];
    __syncthreads();

    // BubbleSort en shared memory
    for (i = 0; i < BLOCK_SIZE; i++) {
        for (j = i + 1; j < BLOCK_SIZE; j++) {
            if (sharedArray[i] > sharedArray[j]) {
                int temp = sharedArray[i];
                sharedArray[i] = sharedArray[j];
                sharedArray[j] = temp;
            }
        }
    }

    // Copiar datos ordenados de vuelta a memoria global
    array[blockIdx.x * BLOCK_SIZE + threadIdx.x] = sharedArray[threadIdx.x];
}

int main() {
    int i;
    int *array, *d_array;

    // Inicializar datos
    array = (int*)malloc(N * sizeof(int));
    for (i = 0; i < N; i++) {
        array[i] = rand() % 100;
    }

    // Copiar datos a la memoria de la GPU
    hipMalloc(&d_array, N * sizeof(int));
    hipMemcpy(d_array, array, N * sizeof(int), hipMemcpyHostToDevice);

    // Ejecutar kernel para ordenar los datos
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    bubbleSort<<<numBlocks, BLOCK_SIZE>>>(d_array);
    hipDeviceSynchronize();

    // Copiar datos ordenados de vuelta a memoria principal
    hipMemcpy(array, d_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // Imprimir los datos ordenados
    for (i = 0; i < N; i++) {
        printf("%d ", array[i]);
    }
    printf("\n");

    // Liberar memoria
    free(array);
    hipFree(d_array);

    return 0;
}
