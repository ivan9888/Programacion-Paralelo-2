#include "hip/hip_runtime.h"
// Streams add vector

include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void stream_add_vec(int* a, int* b, int* c, int size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }
}

int main()
{
    int size = 1 << 18;
    int byte_size = size * sizeof(int);

    int const N_STREAMS = 8;
    int ELEMENTS_PER_STREAM = size / N_STREAMS;
    int BYTES_PER_STREAM = byte_size / N_STREAMS;

    // Initialize host pointer
    int* h_a, * h_b, * h_c;

    hipHostMalloc((void**)&h_a, byte_size);
    hipHostMalloc((void**)&h_b, byte_size);
    hipHostMalloc((void**)&h_c, byte_size);

    // srand((double)time(NULL));
    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // allocate device pointers
    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, byte_size);
    hipMalloc((void**)&d_b, byte_size);
    hipMalloc((void**)&d_c, byte_size);

    // kernel launch
    dim3 block(128);
    dim3 grid(ELEMENTS_PER_STREAM / block.x + 1);

    hipStream_t streams[N_STREAMS];
    for (int i = 0; i < N_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    int offset = 0;

    // trasfer data from host to device
    for (int i = 0; i < N_STREAMS; i++) {
        offset = i * ELEMENTS_PER_STREAM;
        hipMemcpyAsync(d_a + offset, h_a + offset, BYTES_PER_STREAM, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_b + offset, h_b + offset, BYTES_PER_STREAM, hipMemcpyHostToDevice, streams[i]);
        stream_add_vec << <grid, block, 0, streams[i] >> > (d_a + offset, d_b + offset, d_c + offset, ELEMENTS_PER_STREAM);
        hipMemcpyAsync(h_c + offset, d_c + offset, BYTES_PER_STREAM, hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();

    /*for (int i = 0; i < size; i++) {
        printf("\n%d + %d = %d", h_a[i], h_b[i], h_c[i]);
    }*/
   
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}